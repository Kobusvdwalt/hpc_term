#include "hip/hip_runtime.h"
#include "kernel.h"

__host__ __device__
bool Match(float* set1, float* set2, int count) {
	int max1 = 0;
	int max2 = 0;
	for (int i = 0; i < count; i++) {
		if (set1[max1] < set1[i]) {
			max1 = i;
		}
		if (set2[max2] < set2[i]) {
			max2 = i;
		}
	}


	if (max1 == max2) {
		return true;
	}
	return false;
}

__global__ void TrainKernel(GPUData gpuData, Network** networkList) {
	int networkId = blockIdx.x * blockDim.x + threadIdx.x;

	float output[10];

	float* sampleImage = gpuData.trainImages + 784 * networkId;
	float* sampleLabel = gpuData.trainLabels + 10 * networkId;

	for (int j = 0; j < 5; j++) {
		networkList[networkId]->Forward(sampleImage, output);
		networkList[networkId]->Backward(sampleLabel);
	}
	networkList[networkId]->UpdateWeights();
}

void LaunchKernel(dim3 block, dim3 grid, GPUData gpuData, Network** networkList) {
	TrainKernel <<<block, grid >>> (gpuData, networkList);
}

void PrepareGPU(float** trainImages, float** trainLabels, int trainCount, float** testImages, float** testLabels, int testCount, int width, int height) {
	
	// Transfer training data to GPU
	GPUData gpuData = GPUData();
	int trainImagesByteCount = width * height * trainCount * sizeof(float);
	int trainLabelsByteCount = 10 * trainCount * sizeof(float);

	CheckCuda(hipMalloc(&gpuData.trainImages, size_t(trainImagesByteCount)));
	CheckCuda(hipMalloc(&gpuData.trainLabels, size_t(trainLabelsByteCount)));
	gpuData.trainCount = trainCount;

	for (int i = 0; i < trainCount; i++) {
		hipMemcpy(&gpuData.trainImages[i*(width*height)], trainImages[i], width*height * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(&gpuData.trainLabels[i*(10)], trainLabels[i], 10 * sizeof(float), hipMemcpyHostToDevice);
	}
	hipDeviceSynchronize();

	// Create collection of networks
	
	Network* networkCPU = new Network();
	int lc = 3;
	networkCPU->layerCount = new int(3);
	networkCPU->layerWidth = new int[3]{ 784, 64, 10 };
	networkCPU->neuronCount = new int(784 + 64 + 10);
	networkCPU->weightCount = new int(784 * 64 + 64 * 10);

	networkCPU->inputs = new float[*networkCPU->neuronCount * sizeof(float)];
	networkCPU->outputs = new float[*networkCPU->neuronCount * sizeof(float)];
	networkCPU->weights = new float[*networkCPU->weightCount * sizeof(float)];
	networkCPU->biases = new float[*networkCPU->neuronCount * sizeof(float)];

	networkCPU->inputError = new float[*networkCPU->neuronCount * sizeof(float)];
	networkCPU->outputError = new float[*networkCPU->neuronCount * sizeof(float)];
	networkCPU->weightError = new float[*networkCPU->weightCount * sizeof(float)];
	networkCPU->biasError = new float[*networkCPU->neuronCount * sizeof(float)];

	networkCPU->Initialize();
	
	// Data
	int numberOfNetworks = 8192;
	Network** networkList = new Network*[numberOfNetworks];
	
	for (int i = 0; i < numberOfNetworks; i++) {
		Network* network = new Network();
		// Aloocate GPU		
		CheckCuda(hipMalloc(&network->layerCount, sizeof(int)));
		CheckCuda(hipMalloc(&network->neuronCount, sizeof(int)));
		CheckCuda(hipMalloc(&network->weightCount, sizeof(int)));
		CheckCuda(hipMalloc(&network->layerWidth, *networkCPU->layerCount * sizeof(int)));

		CheckCuda(hipMalloc(&network->inputs, *networkCPU->neuronCount * sizeof(float)));
		CheckCuda(hipMalloc(&network->outputs, *networkCPU->neuronCount * sizeof(float)));
		CheckCuda(hipMalloc(&network->weights, *networkCPU->weightCount * sizeof(float)));
		CheckCuda(hipMalloc(&network->biases, *networkCPU->neuronCount * sizeof(float)));

		CheckCuda(hipMalloc(&network->inputError, *networkCPU->neuronCount * sizeof(float)));
		CheckCuda(hipMalloc(&network->outputError, *networkCPU->neuronCount * sizeof(float)));
		CheckCuda(hipMalloc(&network->weightError, *networkCPU->weightCount * sizeof(float)));
		CheckCuda(hipMalloc(&network->biasError, *networkCPU->neuronCount * sizeof(float)));

		// Copy CPU to GPU
		CheckCuda(hipMemcpy(network->layerCount, networkCPU->layerCount, sizeof(int), hipMemcpyHostToDevice));
		CheckCuda(hipMemcpy(network->neuronCount, networkCPU->neuronCount, sizeof(int), hipMemcpyHostToDevice));
		CheckCuda(hipMemcpy(network->weightCount, networkCPU->weightCount, sizeof(int), hipMemcpyHostToDevice));
		CheckCuda(hipMemcpy(network->layerWidth, networkCPU->layerWidth, *networkCPU->layerCount * sizeof(int), hipMemcpyHostToDevice));

		CheckCuda(hipMemcpy(network->inputs, networkCPU->inputs, *networkCPU->neuronCount * sizeof(float), hipMemcpyHostToDevice));
		CheckCuda(hipMemcpy(network->outputs, networkCPU->outputs, *networkCPU->neuronCount * sizeof(float), hipMemcpyHostToDevice));
		CheckCuda(hipMemcpy(network->weights, networkCPU->weights, *networkCPU->weightCount * sizeof(float), hipMemcpyHostToDevice));
		CheckCuda(hipMemcpy(network->biases, networkCPU->biases, *networkCPU->neuronCount * sizeof(float), hipMemcpyHostToDevice));

		CheckCuda(hipMemcpy(network->inputError, networkCPU->inputError, *networkCPU->neuronCount * sizeof(float), hipMemcpyHostToDevice));
		CheckCuda(hipMemcpy(network->outputError, networkCPU->outputError, *networkCPU->neuronCount * sizeof(float), hipMemcpyHostToDevice));
		CheckCuda(hipMemcpy(network->weightError, networkCPU->weightError, *networkCPU->weightCount * sizeof(float), hipMemcpyHostToDevice));
		CheckCuda(hipMemcpy(network->biasError, networkCPU->biasError, *networkCPU->neuronCount * sizeof(float), hipMemcpyHostToDevice));

		// Pointer
		Network* networkPtr;
		CheckCuda(hipMalloc(&networkPtr, sizeof(Network)));
		CheckCuda(hipMemcpy(networkPtr, network, sizeof(Network), hipMemcpyHostToDevice));
		networkList[i] = networkPtr;
	}

	Network** networkListPtr;
	CheckCuda(hipMalloc(&networkListPtr, numberOfNetworks * sizeof(Network*)));
	CheckCuda(hipMemcpy(networkListPtr, networkList, numberOfNetworks * sizeof(Network*), hipMemcpyHostToDevice));

	dim3 blockSize(256, 1);
	dim3 gridSize(numberOfNetworks / 256, 1, 1);

	for (int e = 0; e < 50; e++) {
		double error = 0;
		for (int i = 0; i < trainCount; i++) {
			LaunchKernel(blockSize, gridSize, gpuData, networkListPtr);
			hipDeviceSynchronize();
			printf("Epoch : %d\n", e);
		}
	}

	hipFree(gpuData.trainImages);
	hipFree(gpuData.trainLabels);
	
}

// Serial test code :

/*

		printf("Epoch : %d\n", e);
		printf("Train Error : %lf\n", error / trainCount);

		error = 0;
		for (int i = 0; i < testCount; i++) {
			networkHost.Forward(testImages[i], output);
			if (Match(testLabels[i], output, 10) == false) {
				error += 1.0;
			}
		}
		printf("Test Error : %lf\n", error / testCount);
*/