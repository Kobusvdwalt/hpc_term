#include "hip/hip_runtime.h"
#pragma once
#pragma once
#include "DropoutGPU.h"

__global__
void InitializeKernel(hiprandState* state, int inputWidth) {
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId >= inputWidth) return;
	hiprand_init(0, threadId, 0, &state[threadId]);
}

__global__
void ForwardKernelDropout(hiprandState* randomStates, float dropoutRate, float* input, float* output, float* dropoutMask, int inputWidth) {
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId >= inputWidth) return;

	hiprandState_t state = randomStates[threadId];
	float rand = hiprand_uniform(&state);
	randomStates[threadId] = state;

	float inputR = input[threadId];

	float mask = 0;
	if (rand > dropoutRate) mask = 1;

	dropoutMask[threadId] = mask;
	output[threadId] = inputR * (1.0 / dropoutRate) * mask;
}

__global__
void BackwardKernelDropout(float* inputRef, float* upstreamGradient, float* gradient, float* dropoutMask, float dropoutRate, int inputWidth) {
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId >= inputWidth) return;

	gradient[threadId] = dropoutMask[threadId] * inputRef[threadId] * upstreamGradient[threadId] * (1.0 / dropoutRate);
}

DropoutGPU::DropoutGPU(int inputWidth, float dropoutRate) {
	this->inputWidth = inputWidth;
	this->dropoutRate = dropoutRate;
	CheckCuda(hipMalloc(&inputRef, sizeof(float*)));
	CheckCuda(hipMalloc(&output, inputWidth * sizeof(float)));
	CheckCuda(hipMalloc(&gradient, inputWidth * sizeof(float)));
	CheckCuda(hipMalloc(&dropoutMask, inputWidth * sizeof(float)));

	hipMalloc(&randomStates, inputWidth * sizeof(hiprandState));
	dim3 blockSize(BLOCK_SIZE, 1, 1);
	dim3 gridSize(GRID_SIZE, 1, 1);
	InitializeKernel <<<blockSize, gridSize >>> (randomStates, inputWidth);
}

void DropoutGPU::Forward(float* input) {
	dim3 blockSize(BLOCK_SIZE, 1, 1);
	dim3 gridSize(GRID_SIZE, 1, 1);
	ForwardKernelDropout <<<blockSize, gridSize >>> (randomStates, dropoutRate, input, output, dropoutMask, inputWidth);

	inputRef = input;
}

void DropoutGPU::Backward(float* upstreamGradient) {
	dim3 blockSize(BLOCK_SIZE, 1);
	dim3 gridSize(GRID_SIZE, 1, 1);
	BackwardKernelDropout <<<blockSize, gridSize >>> (inputRef, upstreamGradient, gradient, dropoutMask, dropoutRate, inputWidth);
}